#include "hip/hip_runtime.h"
// Copyright 2019 Open Source Robotics Foundation
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gazebo_plugins/fill_depth.hpp>

#include <iostream>
#include <limits>

class FillDepthPrivate
{
public:
  struct parameters
  {
    /// Image height
    int height;

    /// Image width
    int width;

    /// Size of single image channel
    int size;

    /// Camera focal length
    double fl;

    /// Minimum depth value in image
    double min_depth;

    /// Maximum depth value in image
    double max_depth;

    /// Value of infinity
    float infinity;
  } params;

  /// Pointer to struct parameters
  parameters * gpu_params;

  /// Pointer to depth information stored in GPU
  float * gpu_image_depth;

  /// Pointer to RGB image stored in GPU
  uint8_t * gpu_image_rgb;

  /// Pointer to pointcloud computed in GPU
  float * gpu_cloud;

  /// Pointer to depth image computed in GPU
  float * gpu_depth;

  /// Number of GPU threads to use
  int threads;

  /// Number of GPU blocks to use
  int blocks;

  /// Size of single image channel
  int size;
};

extern "C" __global__ void fill(
  FillDepthPrivate::parameters * params,
  float * image_depth, uint8_t * image_rgb, float * depth, float * cloud)
{
  double pAngle = 0;
  double yAngle = 0;

  typedef union {
    uint8_t rgb_int[3];
    float rgb_float;
  } rgb_data;
  rgb_data data;

  for (uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    index < params->size; index += blockDim.x * gridDim.x)
  {
    uint32_t i = index % params->width;
    uint32_t j = index / params->width;

    if (params->height > 1) {
      pAngle = atan2(
        static_cast<double>(j) - 0.5 * static_cast<double>(params->height - 1), params->fl);
    }

    if (params->width > 1) {
      yAngle = atan2(
        static_cast<double>(i) - 0.5 * static_cast<double>(params->width - 1), params->fl);
    }
    // in optical frame
    // hardcoded rotation rpy(-M_PI/2, 0, -M_PI/2) is built-in
    // to urdf, where the *_optical_frame should have above relative
    // rotation from the physical camera *_frame

    float distance = image_depth[index];

    if (distance > params->min_depth && distance < params->max_depth) {
      depth[index] = distance;

      auto x = static_cast<float>(distance * tan(yAngle));
      auto y = static_cast<float>(distance * tan(pAngle));

      cloud[index * 8] = x;
      cloud[index * 8 + 1] = y;
      cloud[index * 8 + 2] = distance;
    } else if (distance <= params->min_depth) {
      depth[index] = -params->infinity;

      cloud[index * 8] = -params->infinity;
      cloud[index * 8 + 1] = -params->infinity;
      cloud[index * 8 + 2] = -params->infinity;
    } else {
      depth[index] = params->infinity;

      cloud[index * 8] = params->infinity;
      cloud[index * 8 + 1] = params->infinity;
      cloud[index * 8 + 2] = params->infinity;
    }

    data.rgb_int[0] = image_rgb[index * 3];
    data.rgb_int[1] = image_rgb[index * 3 + 1];
    data.rgb_int[2] = image_rgb[index * 3 + 2];

    cloud[index * 8 + 4] = data.rgb_float;
  }
}

FillDepth::FillDepth()
{
  impl_ = (FillDepthPrivate *) std::malloc(sizeof(FillDepthPrivate));
  hipMalloc(reinterpret_cast<void **>(&(impl_->gpu_params)), sizeof(impl_->params));
}

FillDepth::~FillDepth()
{
  hipFree(impl_->gpu_params);
  hipFree(impl_->gpu_image_depth);
  hipFree(impl_->gpu_image_rgb);
  hipFree(impl_->gpu_cloud);
  free(impl_);
}

void FillDepth::initialize(
  unsigned int height, unsigned int width, double fl, double min_depth, double max_depth)
{
  impl_->params.height = height;
  impl_->params.width = width;
  impl_->params.fl = fl;
  impl_->params.min_depth = min_depth;
  impl_->params.max_depth = max_depth;
  impl_->size = impl_->params.width * impl_->params.height;
  impl_->params.size = impl_->size;
  impl_->params.infinity = std::numeric_limits<float>::infinity();

  hipMemcpy(impl_->gpu_params, &impl_->params, sizeof(impl_->params), hipMemcpyHostToDevice);
  hipMalloc(reinterpret_cast<void **>(&(impl_->gpu_image_depth)), impl_->size * sizeof(float));
  hipMalloc(reinterpret_cast<void **>(&(impl_->gpu_image_rgb)), 3 * impl_->size * sizeof(uint8_t));
  hipMalloc(reinterpret_cast<void **>(&(impl_->gpu_depth)), impl_->size * sizeof(float));
  hipMalloc(reinterpret_cast<void **>(&(impl_->gpu_cloud)), 8 * impl_->size * sizeof(float));

  impl_->threads = 512;
  impl_->blocks = (impl_->size + impl_->threads - 1) / impl_->threads;
}

void FillDepth::compute(
  const float * image_depth, uint8_t * image_rgb, float * depth, float * cloud)
{
  hipMemcpy(
    impl_->gpu_image_depth, image_depth, impl_->size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(
    impl_->gpu_image_rgb, image_rgb, 3 * impl_->size * sizeof(uint8_t), hipMemcpyHostToDevice);

  fill <<< impl_->blocks, impl_->threads >>> (
    impl_->gpu_params, impl_->gpu_image_depth, impl_->gpu_image_rgb,
    impl_->gpu_depth, impl_->gpu_cloud);

  hipMemcpy(depth, impl_->gpu_depth, impl_->size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(cloud, impl_->gpu_cloud, 8 * impl_->size * sizeof(float), hipMemcpyDeviceToHost);
}
